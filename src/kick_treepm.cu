#include "hip/hip_runtime.h"
#include <tigerpm/kick_treepm.hpp>
#include <tigerpm/timer.hpp>
#include <tigerpm/options.hpp>
#include <tigerpm/particles.hpp>
#include <tigerpm/util.hpp>
#include <tigerpm/gravity_long.hpp>
#include <tigerpm/gravity_short.hpp>

#include <thrust/device_vector.h>

#include <algorithm>

struct fixed4 {
	fixed32 x;
	fixed32 y;
	fixed32 z;
	float m;
};

__constant__ float rung_dt[MAX_RUNG] = { 1.0 / (1 << 0), 1.0 / (1 << 1), 1.0 / (1 << 2), 1.0 / (1 << 3), 1.0 / (1 << 4), 1.0 / (1 << 5), 1.0 / (1 << 6), 1.0
		/ (1 << 7), 1.0 / (1 << 8), 1.0 / (1 << 9), 1.0 / (1 << 10), 1.0 / (1 << 11), 1.0 / (1 << 12), 1.0 / (1 << 13), 1.0 / (1 << 14), 1.0 / (1 << 15), 1.0
		/ (1 << 16), 1.0 / (1 << 17), 1.0 / (1 << 18), 1.0 / (1 << 19), 1.0 / (1 << 20), 1.0 / (1 << 21), 1.0 / (1 << 22), 1.0 / (1 << 23), 1.0 / (1 << 24), 1.0
		/ (1 << 25), 1.0 / (1 << 26), 1.0 / (1 << 27), 1.0 / (1 << 28), 1.0 / (1 << 29), 1.0 / (1 << 30), 1.0 / (1 << 31) };

struct sink_cell {
	int begin;
	int end;
	array<int, NDIM> loc;
};

#define WORKSPACE_SIZE  1024
#define INTERSPACE_SIZE (2 * TREEPM_BLOCK_SIZE * SOURCE_BUCKET_SIZE)

struct treepm_params {
	fixed32* x;
	fixed32* y;
	fixed32* z;
	float* velx;
	float* vely;
	float* velz;
	float* phi;
	char* rung;
	int* checklist;
	int* nextlist;
	fixed4* sourcelist;
	tree* tree_neighbors;
	sink_bucket** buckets;
	int* bucket_cnt;
	int nsink_cells;
	int min_rung;
	float rs;
	float GM;
	float eta;
	float t0;
	float scale;
	float hsoft;
	float inv2rs;
	float twooversqrtpi;
	float h2;
	float hinv;
	float h3inv;
	float theta;
	bool first_call;
	int Nfour;
	range<int> phi_box;
#ifdef FORCE_TEST
	float* gx;
	float* gy;
	float* gz;
	float* pot;
#endif
	void allocate(size_t source_size, size_t sink_size, size_t cell_count, size_t big_cell_count, size_t phi_cell_count, int nblocks) {
		nsink_cells = cell_count;
		CUDA_CHECK(hipMalloc(&tree_neighbors, cell_count * NCELLS * sizeof(tree)));
		CUDA_CHECK(hipMalloc(&buckets, cell_count * sizeof(sink_bucket*)));
		CUDA_CHECK(hipMalloc(&bucket_cnt, cell_count * sizeof(int)));
		CUDA_CHECK(hipMalloc(&x, source_size * sizeof(fixed32)));
		CUDA_CHECK(hipMalloc(&y, source_size * sizeof(fixed32)));
		CUDA_CHECK(hipMalloc(&z, source_size * sizeof(fixed32)));
		CUDA_CHECK(hipMalloc(&velx, sink_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&vely, sink_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&velz, sink_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&rung, sink_size * sizeof(char)));
		CUDA_CHECK(hipMalloc(&phi, phi_cell_count * sizeof(float)));
		CUDA_CHECK(hipMalloc(&checklist, nblocks * sizeof(int) * WORKSPACE_SIZE));
		CUDA_CHECK(hipMalloc(&nextlist, nblocks * sizeof(int) * WORKSPACE_SIZE));
		CUDA_CHECK(hipMalloc(&sourcelist, nblocks * sizeof(fixed4) * INTERSPACE_SIZE));
#ifdef FORCE_TEST
		CUDA_CHECK(hipMalloc(&gx, source_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&gy, source_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&gz, source_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&pot, source_size * sizeof(float)));
#endif
	}
	void free() {
		CUDA_CHECK(hipFree(x));
		CUDA_CHECK(hipFree(y));
		CUDA_CHECK(hipFree(z));
		CUDA_CHECK(hipFree(velx));
		CUDA_CHECK(hipFree(vely));
		CUDA_CHECK(hipFree(velz));
		CUDA_CHECK(hipFree(buckets));
		CUDA_CHECK(hipFree(bucket_cnt));
		CUDA_CHECK(hipFree(phi));
		CUDA_CHECK(hipFree(rung));
		CUDA_CHECK(hipFree(tree_neighbors));
		CUDA_CHECK(hipFree(checklist));
		CUDA_CHECK(hipFree(nextlist));
		CUDA_CHECK(hipFree(sourcelist));
#ifdef FORCE_TEST
		CUDA_CHECK(hipFree(gx));
		CUDA_CHECK(hipFree(gy));
		CUDA_CHECK(hipFree(gz));
		CUDA_CHECK(hipFree(pot));
#endif
	}
};

static size_t mem_requirements(int nsources, int nsinks, int vol, int bigvol, int phivol) {
	size_t mem = 0;
	mem += NDIM * sizeof(fixed32) * nsources;
	mem += NDIM * sizeof(float) * nsinks;
	mem += sizeof(char) * nsinks;
	mem += NCELLS * bigvol * sizeof(tree);
	mem += vol * sizeof(sink_cell);
	mem += 2 * sizeof(int) * vol;
	mem += sizeof(sink_bucket*) * vol;
	mem += sizeof(int) * vol;
	mem += sizeof(tree) * vol;
	mem += phivol * sizeof(float);
	mem += sizeof(treepm_params);
#ifdef FORCE_TEST
	mem += (NDIM + 1) * sizeof(float) * nsinks;
#endif
	return mem;
}

struct cpymem {
	void* dest;
	void* src;
	size_t size;
};

static void process_copies(vector<cpymem> copies, hipMemcpyKind direction, hipStream_t stream) {
	vector<cpymem> compressed;
	std::sort(copies.begin(), copies.end(), [](cpymem a, cpymem b) {
		return a.dest < b.dest;
	});
	for (int i = 0; i < copies.size(); i++) {
		cpymem copy = copies[i];
		for (int j = i + 1; j < copies.size(); j++) {
			if (((char*) copy.dest + copy.size == copies[j].dest) && ((char*) copy.src + copy.size == copies[j].src)) {
				copy.size += copies[j].size;
				i++;
			} else {
				break;
			}
		}
		compressed.push_back(copy);
	}
	PRINT("Compressed from %i to %i copies\n", copies.size(), compressed.size());
	for (int i = 0; i < compressed.size(); i++) {
		CUDA_CHECK(hipMemcpyAsync(compressed[i].dest, compressed[i].src, compressed[i].size, direction, stream));
	}
}

struct treepm_shmem {
	array<array<float, NDIM>, SINK_BUCKET_SIZE> g;
	array<float, SINK_BUCKET_SIZE> phi;
	array<int, SINK_BUCKET_SIZE> active_srci;
	array<int, SINK_BUCKET_SIZE> active_snki;
	array<float, TREEPM_BLOCK_SIZE> reduce;
	array<int, TREEPM_BLOCK_SIZE> index;
	array<int, TREEPM_BLOCK_SIZE> partlist;
};

__constant__ treepm_params dev_treepm_params;

__device__ int compute_indices(array<int, TREEPM_BLOCK_SIZE>& index) {
	const int& tid = threadIdx.x;
	for (int P = 1; P < TREEPM_BLOCK_SIZE; P *= 2) {
		int tmp;
		__syncthreads();
		if (tid >= P) {
			tmp = index[tid - P];
		}
		__syncthreads();
		if (tid >= P) {
			index[tid] += tmp;
		}
	}
	__syncthreads();
	return (tid > 0 ? index[tid - 1] : 0);

}

__global__ void kick_treepm_kernel() {
	const treepm_params& params = dev_treepm_params;
	__shared__ treepm_shmem shmem;
	const int& tid = threadIdx.x;
	const int& bid = blockIdx.x;
	const int& gsz = gridDim.x;
	const float& inv2rs = params.inv2rs;
	const float& twooversqrtpi = params.twooversqrtpi;
	const float& h2 = params.h2;
	const float& hinv = params.hinv;
	const float& h3inv = params.h3inv;
	const float rcut = 5.f * params.rs;
	const float theta2inv = 1.0f / sqr(params.theta);
	const int cell_begin = size_t(bid) * (size_t) params.nsink_cells / (size_t) gsz;
	const int cell_end = size_t(bid + 1) * (size_t) params.nsink_cells / (size_t) gsz;
	for (int cell_index = cell_begin; cell_index < cell_end; cell_index++) {
		sink_bucket* buckets = params.buckets[cell_index];
		const int& bucket_cnt = params.bucket_cnt[cell_index];
		for (int bi = 0; bi < bucket_cnt; bi++) {
			const auto& bucket = buckets[bi];
			const auto& snk_begin = bucket.snk_begin;
			const auto& snk_end = bucket.snk_end;
			__syncthreads();
			const int nsinks = snk_end - snk_begin;
			const int imax = round_up(nsinks, TREEPM_BLOCK_SIZE);
			int nactive = 0;
			for (int i = tid; i < imax; i += TREEPM_BLOCK_SIZE) {
				const int this_index = snk_begin + i;
				bool is_active;
				if (i < nsinks) {
					is_active = int(params.rung[this_index] >= params.min_rung);
				} else {
					is_active = false;
				}
				shmem.index[tid] = int(is_active);
				int active_index = compute_indices(shmem.index) + +nactive;
				if (is_active) {
					shmem.active_snki[active_index] = this_index;
					shmem.active_srci[active_index] = bucket.src_begin + i;
				}
				nactive += shmem.index[TREEPM_BLOCK_SIZE - 1];
				__syncthreads();
			}
			for (int sink_index = tid; sink_index < nactive; sink_index += TREEPM_BLOCK_SIZE) {
				array<float, NDIM>& g = shmem.g[sink_index];
				float& phi = shmem.phi[sink_index];
				g[0] = g[1] = g[2] = 0.f;
				phi = 0.f;
				const int srci = shmem.active_srci[sink_index];
				const fixed32 sink_x = params.x[srci];
				const fixed32 sink_y = params.y[srci];
				const fixed32 sink_z = params.z[srci];
				array<int, NDIM> I;
				array<int, NDIM> J;
				array<float, NDIM> X;
				X[XDIM] = sink_x.to_float();
				X[YDIM] = sink_y.to_float();
				X[ZDIM] = sink_z.to_float();
				array<array<float, NINTERP>, NINTERP> w;
				array<array<float, NINTERP>, NINTERP> dw;
				for (int dim = 0; dim < NDIM; dim++) {
					X[dim] *= params.Nfour;
					I[dim] = min(int(X[dim]), params.phi_box.end[dim] - PHI_BW);
					X[dim] -= float(I[dim]);
					I[dim] -= 2;
				}
				for (int dim = 0; dim < NDIM; dim++) {
					float x1 = X[dim];
					float x2 = X[dim] * x1;
					float x3 = x1 * x2;
					float x4 = x2 * x2;
					float x5 = x3 * x2;
					w[dim][0] = (1.f / 12.f) * x1 - (1.f / 24.f) * x2 - (3.f / 8.f) * x3 + (13.f / 24.f) * x4 - (5.f / 24.f) * x5;
					w[dim][1] = -(2.f / 3.f) * x1 + (2.f / 3.f) * x2 + (13.f / 8.f) * x3 - (8.f / 3.f) * x4 + (25.f / 24.f) * x5;
					w[dim][2] = 1.0f - (5.f / 4.f) * x2 - (35.f / 12.f) * x3 + (21.f / 4.f) * x4 - (25.f / 12.f) * x5;
					w[dim][3] = (2.f / 3.f) * x1 + (2.f / 3.f) * x2 + (11.f / 4.f) * x3 - (31.f / 6.f) * x4 + (25.f / 12.f) * x5;
					w[dim][4] = -(1.f / 12.f) * x1 - (1.f / 24.f) * x2 - (11.f / 8.f) * x3 + (61.f / 24.f) * x4 - (25.f / 24.f) * x5;
					w[dim][5] = (7.f / 24.f) * x3 - (0.5f) * x4 + (5.f / 24.f) * x5;
					x5 = 5.0f * x4;
					x4 = 4.0f * x3;
					x3 = 3.0f * x2;
					x2 = 2.0f * x1;
					x1 = 1.0f;
					dw[dim][0] = (1.f / 12.f) * x1 - (1.f / 24.f) * x2 - (3.f / 8.f) * x3 + (13.f / 24.f) * x4 - (5.f / 24.f) * x5;
					dw[dim][1] = -(2.f / 3.f) * x1 + (2.f / 3.f) * x2 + (13.f / 8.f) * x3 - (8.f / 3.f) * x4 + (25.f / 24.f) * x5;
					dw[dim][2] = -(5.f / 4.f) * x2 - (35.f / 12.f) * x3 + (21.f / 4.f) * x4 - (25.f / 12.f) * x5;
					dw[dim][3] = (2.f / 3.f) * x1 + (2.f / 3.f) * x2 + (11.f / 4.f) * x3 - (31.f / 6.f) * x4 + (25.f / 12.f) * x5;
					dw[dim][4] = -(1.f / 12.f) * x1 - (1.f / 24.f) * x2 - (11.f / 8.f) * x3 + (61.f / 24.f) * x4 - (25.f / 24.f) * x5;
					dw[dim][5] = (7.f / 24.f) * x3 - (0.5f) * x4 + (5.f / 24.f) * x5;
				}
				for (int dim1 = 0; dim1 < NDIM; dim1++) {
					for (J[0] = I[0]; J[0] < I[0] + NINTERP; J[0]++) {
						for (J[1] = I[1]; J[1] < I[1] + NINTERP; J[1]++) {
							for (J[2] = I[2]; J[2] < I[2] + NINTERP; J[2]++) {
								double w0 = 1.0;
								for (int dim2 = 0; dim2 < NDIM; dim2++) {
									const int i0 = J[dim2] - I[dim2];
									if (dim1 == dim2) {
										w0 *= dw[dim2][i0];
									} else {
										w0 *= w[dim2][i0];
									}
								}
								const int l = params.phi_box.index(J);
								g[dim1] -= w0 * params.phi[l] * params.Nfour;
							}
						}
					}
				}
				for (J[0] = I[0]; J[0] < I[0] + NINTERP; J[0]++) {
					for (J[1] = I[1]; J[1] < I[1] + NINTERP; J[1]++) {
						for (J[2] = I[2]; J[2] < I[2] + NINTERP; J[2]++) {
							double w0 = 1.0;
							for (int dim2 = 0; dim2 < NDIM; dim2++) {
								const int i0 = J[dim2] - I[dim2];
								w0 *= w[dim2][i0];
							}
							const int l = params.phi_box.index(J);
							phi += w0 * params.phi[l];
						}
					}
				}
			}
			const size_t offset = bid * WORKSPACE_SIZE;
			int* checklist = params.checklist + offset;
			int* nextlist = params.nextlist + offset;
			fixed4* sourcelist = params.sourcelist + bid * INTERSPACE_SIZE;
			const auto& sink_x = bucket.x[XDIM];
			const auto& sink_y = bucket.x[YDIM];
			const auto& sink_z = bucket.x[ZDIM];
			const auto& sink_radius = bucket.radius;
			for (int tree_index = 0; tree_index < NCELLS; tree_index++) {
				tree& tr = params.tree_neighbors[cell_index * NCELLS + tree_index];
				int check_size = 1;
				int next_size = 0;
				int source_size = 0;
				checklist[0] = 0;
				const auto process_sources = [&]() {
					for (int sink_index = tid; sink_index < nactive; sink_index += TREEPM_BLOCK_SIZE) {
						float& phi = shmem.phi[sink_index];
						auto& g = shmem.g[sink_index];
						const int srci = shmem.active_srci[sink_index];
						const fixed32 sink_x = params.x[srci];
						const fixed32 sink_y = params.y[srci];
						const fixed32 sink_z = params.z[srci];
						for (int i = 0; i < source_size; i++) {
							const fixed32& src_x = sourcelist[i].x;
							const fixed32& src_y = sourcelist[i].y;
							const fixed32& src_z = sourcelist[i].z;
							const float& m = sourcelist[i].m;
							const float dx = distance(sink_x, src_x);
							const float dy = distance(sink_y, src_y);
							const float dz = distance(sink_z, src_z);
							const float r2 = sqr(dx, dy, dz);
							float rinv, rinv3;
							if (r2 > h2) {
								const float r = sqrtf(r2);
								rinv = rsqrtf(r2);
								const float r0 = r * inv2rs;
								float exp0;
								const float erfc0 = erfcexp(r0, &exp0);
								rinv3 = (erfc0 + twooversqrtpi * r0 * exp0) * rinv * rinv * rinv;
								rinv *= erfc0;
							} else {
								const float q = sqrtf(r2) * hinv;
								const float q2 = q * q;
								rinv3 = +15.0f / 8.0f;
								rinv3 = fmaf(rinv3, q2, -21.0f / 4.0f);
								rinv3 = fmaf(rinv3, q2, +35.0f / 8.0f);
								rinv3 *= h3inv;
								rinv = -5.0f / 16.0f;
								rinv = fmaf(rinv, q2, 21.0f / 16.0f);
								rinv = fmaf(rinv, q2, -35.0f / 16.0f);
								rinv = fmaf(rinv, q2, 35.0f / 16.0f);
								rinv *= hinv;
							}
							rinv3 *= m;
							rinv *= m;
							g[XDIM] -= dx * rinv3;
							g[YDIM] -= dy * rinv3;
							g[ZDIM] -= dz * rinv3;
							phi -= rinv;
						}
					}
				};
				while (check_size) {
					const int maxi = round_up(check_size, TREEPM_BLOCK_SIZE);
					for (int ci = tid; ci < maxi; ci += TREEPM_BLOCK_SIZE) {
						int this_index = 0;
						bool nextb = false;
						bool multib = false;
						bool partb = false;
						fixed32 source_x;
						fixed32 source_y;
						fixed32 source_z;
						int index;
						if (ci < check_size) {
							index = checklist[ci];
							source_x = tr.get_x(0, index);
							source_y = tr.get_x(1, index);
							source_z = tr.get_x(2, index);
							const int nparts = tr.get_pend(index) - tr.get_pbegin(index);
							const float source_radius = tr.get_radius(index);
							const float dx = distance(sink_x, source_x);
							const float dy = distance(sink_y, source_y);
							const float dz = distance(sink_z, source_z);
							const float R2 = sqr(dx, dy, dz);
							const float R = sqrtf(R2);
							const bool cutoff = R - source_radius - sink_radius > rcut;
							const bool far = R2 > sqr(source_radius + sink_radius) * theta2inv;
							//		PRINT( "%e %e %e\n", R2, source_radius, sink_radius);
							const bool leaf = nparts <= SOURCE_BUCKET_SIZE;
							multib = !cutoff && far;
							partb = !cutoff && !far && leaf;
							nextb = !cutoff && !far && !leaf;
							//		PRINT( "%i %i %i \n", multib, partb, nextb);
						}
						shmem.index[tid] = multib;
						this_index = compute_indices(shmem.index) + source_size;
						if (source_size + shmem.index[TREEPM_BLOCK_SIZE - 1] >= INTERSPACE_SIZE) {
							PRINT("internal interspace exceeded on multipoles\n");
							__trap();
							assert(false);
						}
						if (multib) {
							sourcelist[this_index].x = source_x;
							sourcelist[this_index].y = source_y;
							sourcelist[this_index].z = source_z;
							sourcelist[this_index].m = tr.get_mass(index);
						}
						source_size += shmem.index[TREEPM_BLOCK_SIZE - 1];
						__syncthreads();

						shmem.index[tid] = partb;
						this_index = compute_indices(shmem.index);
						if (partb) {
							shmem.partlist[this_index] = index;
						}
						const int part_size = shmem.index[TREEPM_BLOCK_SIZE - 1];
						__syncthreads();

						for (int j = 0; j < part_size; j++) {
							const int begin = tr.get_pbegin(shmem.partlist[j]);
							const int end = tr.get_pend(shmem.partlist[j]);
							for (int k = begin + tid; k < end; k += TREEPM_BLOCK_SIZE) {
								const int l = source_size + k - begin;
								sourcelist[l].x = params.x[k];
								sourcelist[l].y = params.y[k];
								sourcelist[l].z = params.z[k];
								sourcelist[l].m = 1.f;
							}
							source_size += end - begin;
						}

						shmem.index[tid] = nextb;
						this_index = compute_indices(shmem.index);
						if (next_size + shmem.index[TREEPM_BLOCK_SIZE - 1] >= WORKSPACE_SIZE) {
							PRINT("internal workspace exceeded\n");
							__trap();
							assert(false);
						}
						if (nextb) {
							const auto children = tr.get_children(index);
							nextlist[next_size + 2 * this_index + 0] = children[0];
							nextlist[next_size + 2 * this_index + 1] = children[1];
						}
						next_size += 2 * shmem.index[TREEPM_BLOCK_SIZE - 1];
						__syncthreads();

					}
					auto tmp1 = nextlist;
					nextlist = checklist;
					checklist = tmp1;
					check_size = next_size;
					next_size = 0;
					if (source_size > INTERSPACE_SIZE / 2) {
						process_sources();
						source_size = 0;
					}
				}
				process_sources();
				for (int sink_index = tid; sink_index < nactive; sink_index += TREEPM_BLOCK_SIZE) {
					array<float, NDIM>& g = shmem.g[sink_index];
					float& phi = shmem.phi[sink_index];
					const int snki = shmem.active_snki[sink_index];
					g[XDIM] *= params.GM;
					g[YDIM] *= params.GM;
					g[ZDIM] *= params.GM;
					phi *= params.GM;
#ifdef FORCE_TEST
					params.gx[snki] = g[XDIM];
					params.gy[snki] = g[YDIM];
					params.gz[snki] = g[ZDIM];
					params.pot[snki] = phi;
#endif
					auto& vx = params.velx[snki];
					auto& vy = params.vely[snki];
					auto& vz = params.velz[snki];
					auto& rung = params.rung[snki];
					auto dt = 0.5f * rung_dt[rung] * params.t0;
					if (!params.first_call) {
						vx = fmaf(g[XDIM], dt, vx);
						vy = fmaf(g[YDIM], dt, vy);
						vz = fmaf(g[ZDIM], dt, vz);
					}
					const auto g2 = sqr(g[0], g[1], g[2]);
					const auto factor = params.eta * sqrtf(params.scale * params.hsoft);
					dt = fminf(factor * rsqrt(sqrtf(g2)), params.t0);
					rung = fmaxf(ceilf(log2f(params.t0) - log2f(dt)), rung - 1);
					if (rung < 0 || rung >= MAX_RUNG) {
						PRINT("Rung out of range %i\n", rung);
					}
					assert(rung >= 0);
					assert(rung < MAX_RUNG);
					dt = 0.5f * rung_dt[rung] * params.t0;
					vx = fmaf(g[XDIM], dt, vx);
					vy = fmaf(g[YDIM], dt, vy);
					vz = fmaf(g[ZDIM], dt, vz);
					//		PRINT( "%i\n", snki);
				}
			}
		}
	}
}

void kick_treepm(vector<tree> trees, vector<vector<sink_bucket>> buckets, range<int> box, int min_rung, double scale, double t0, bool first_call) {
	PRINT("Sorting cells\n");
	timer tm;
	size_t nsources = 0;
	size_t nsinks = 0;
	array<int, NDIM> i;
	const auto bigbox = box.pad(1);
	const size_t bigvol = bigbox.volume();
	const size_t vol = box.volume();
	int tree_size = 0;
	int buckets_size = 0;
	print("%i\n", bigvol);
	for (i[0] = bigbox.begin[0]; i[0] != bigbox.end[0]; i[0]++) {
		for (i[1] = bigbox.begin[1]; i[1] != bigbox.end[1]; i[1]++) {
			for (i[2] = bigbox.begin[2]; i[2] != bigbox.end[2]; i[2]++) {
				auto this_cell = chainmesh_get(i);
				nsources += this_cell.pend - this_cell.pbegin;
			}
		}
	}
	for (i[0] = box.begin[0]; i[0] != box.end[0]; i[0]++) {
		for (i[1] = box.begin[1]; i[1] != box.end[1]; i[1]++) {
			for (i[2] = box.begin[2]; i[2] != box.end[2]; i[2]++) {
				auto this_cell = chainmesh_get(i);
				nsinks += this_cell.pend - this_cell.pbegin;
				const int index = box.index(i);
				tree_size += trees[index].size();
				buckets_size += sizeof(sink_bucket) * buckets[index].size();
			}
		}
	}
	auto phibox = box;
	for (int dim = 0; dim < NDIM; dim++) {
		phibox.begin[dim] *= get_options().four_o_chain;
		phibox.end[dim] *= get_options().four_o_chain;
	}
	phibox = phibox.pad(PHI_BW);
	hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(kick_treepm_kernel));
	if (attr.maxThreadsPerBlock < TREEPM_BLOCK_SIZE) {
		PRINT("This CUDA device will not run kick_pme_kernel with the required number of threads (%i)\n", TREEPM_BLOCK_SIZE);
		abort();
	}
	int occupancy;
	CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor ( &occupancy, kick_treepm_kernel,TREEPM_BLOCK_SIZE, sizeof(treepm_shmem)));
	int num_blocks = TREEPM_OVERSUBSCRIPTION * occupancy * cuda_smp_count();
	const size_t mem_required = mem_requirements(nsources, nsinks, vol, bigvol, phibox.volume()) + tree_size + buckets_size
			+ num_blocks * sizeof(int) * 2 * (WORKSPACE_SIZE + INTERSPACE_SIZE);
	const size_t free_mem = (size_t) 85 * cuda_free_mem() / size_t(100);
	PRINT("required = %li freemem = %li\n", mem_required, free_mem);
	if (mem_required > free_mem) {
		const auto child_boxes = box.split();
		PRINT("Splitting\n");
		kick_treepm(trees, buckets, child_boxes.first, min_rung, scale, t0, first_call);
		kick_treepm(trees, buckets, child_boxes.second, min_rung, scale, t0, first_call);
	} else {
		cuda_set_device();
		PRINT("Data transfer\n");
		tm.start();
		treepm_params params;
		params.allocate(nsources, nsinks, vol, bigvol, phibox.volume(), num_blocks);
		tm.stop();
		PRINT("%e\n", tm.read());
		tm.start();
		params.theta = 0.5;
		params.min_rung = min_rung;
		params.rs = get_options().rs;
		params.GM = get_options().GM;
		params.Nfour = get_options().four_dim;
		params.phi_box = phibox;
		params.eta = get_options().eta;
		params.first_call = first_call;
		params.t0 = t0;
		params.scale = scale;
		params.hsoft = get_options().hsoft;
		params.inv2rs = 1.0f / params.rs / 2.0f;
		params.twooversqrtpi = 2.0f / sqrtf(M_PI);
		params.h2 = sqr(params.hsoft);
		params.hinv = 1.f / params.hsoft;
		params.h3inv = params.hinv * sqr(params.hinv);
		tree* dev_tree_neighbors = (tree*) malloc(sizeof(tree) * NCELLS * vol);
		hipStream_t stream;
		CUDA_CHECK(hipStreamCreate(&stream));
		auto phi = gravity_long_get_phi(phibox);
		CUDA_CHECK(hipMemcpyAsync(params.phi, phi.data(), sizeof(float) * phi.size(), hipMemcpyHostToDevice, stream));

		struct cell_data {
			int box_index;
			int bigbox_index;
			chaincell cell;
		};
		vector<cell_data> chaincells;
		for (i[0] = bigbox.begin[0]; i[0] != bigbox.end[0]; i[0]++) {
			for (i[1] = bigbox.begin[1]; i[1] != bigbox.end[1]; i[1]++) {
				for (i[2] = bigbox.begin[2]; i[2] != bigbox.end[2]; i[2]++) {
					cell_data entry;
					entry.bigbox_index = bigbox.index(i);
					entry.cell = chainmesh_get(i);
					if (box.contains(i)) {
						const int q = box.index(i);
						entry.box_index = q;
					} else {
						entry.box_index = -1;
					}
					chaincells.push_back(entry);
				}
			}
		}
		std::sort(chaincells.begin(), chaincells.end(), [](cell_data a, cell_data b) {
			return a.cell.pbegin < b.cell.pbegin;
		});
		size_t count = 0;
		vector<cpymem> copies;
		for (int j = 0; j < chaincells.size(); j++) {
			auto this_cell = chaincells[j].cell;
			const auto this_size = this_cell.pend - this_cell.pbegin;
			const auto begin = this_cell.pbegin;
			const auto dif = count - begin;
			const int l = chaincells[j].bigbox_index;
			trees[l].adjust_indexes(dif);
			if (chaincells[j].box_index >= 0) {
				const int q = chaincells[j].box_index;
				for (auto& bucket : buckets[q]) {
					bucket.src_begin += dif;
					bucket.src_end += dif;
				}
			}
			cpymem cpy;
			cpy.size = sizeof(fixed32) * this_size;
			cpy.dest = params.x + count;
			cpy.src = &particles_pos(XDIM, begin);
			copies.push_back(cpy);
			cpy.dest = params.y + count;
			cpy.src = &particles_pos(YDIM, begin);
			copies.push_back(cpy);
			cpy.dest = params.z + count;
			cpy.src = &particles_pos(ZDIM, begin);
			copies.push_back(cpy);
			count += this_size;
		}
		vector<tree> dev_trees(bigvol);
		for (int j = 0; j < bigvol; j++) {
			dev_trees[j] = trees[j].to_device(stream);
		}
		count = 0;
		for (i[0] = box.begin[0]; i[0] != box.end[0]; i[0]++) {
			for (i[1] = box.begin[1]; i[1] != box.end[1]; i[1]++) {
				for (i[2] = box.begin[2]; i[2] != box.end[2]; i[2]++) {
					auto this_cell = chainmesh_get(i);
					const auto this_size = this_cell.pend - this_cell.pbegin;
					const auto begin = this_cell.pbegin;
					cpymem cpy;
					const int l = box.index(i);
					const auto dif = count - begin;
					for (auto& bucket : buckets[l]) {
						bucket.snk_begin += dif;
						bucket.snk_end += dif;
					}
					cpy.size = sizeof(float) * this_size;
					cpy.dest = params.velx + count;
					cpy.src = &particles_vel(XDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.vely + count;
					cpy.src = &particles_vel(YDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.velz + count;
					cpy.src = &particles_vel(ZDIM, begin);
					copies.push_back(cpy);
					cpy.size = sizeof(char) * this_size;
					cpy.dest = params.rung + count;
					cpy.src = &particles_rung(begin);
					copies.push_back(cpy);
#ifdef FORCE_TEST
					cpy.size = sizeof(float) * this_size;
					cpy.dest = params.gx + count;
					cpy.src = &particles_gforce(XDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.gy + count;
					cpy.src = &particles_gforce(YDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.gz + count;
					cpy.src = &particles_gforce(ZDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.pot + count;
					cpy.src = &particles_pot(begin);
					copies.push_back(cpy);
#endif
					count += this_size;
					array<int, NDIM> j;
					int p = 0;
					for (j[0] = i[0] - 1; j[0] <= i[0] + 1; j[0]++) {
						for (j[1] = i[1] - 1; j[1] <= i[1] + 1; j[1]++) {
							for (j[2] = i[2] - 1; j[2] <= i[2] + 1; j[2]++) {
								const int k = bigbox.index(j);
								std::memcpy(&dev_tree_neighbors[p + NCELLS * l], &dev_trees[k], sizeof(tree));
								p++;
							}
						}
					}
				}
			}
		}
		vector<sink_bucket*> dev_buckets;
		vector<int> bucket_count;
		timer tm1;
		tm1.start();
		for (int j = 0; j < vol; j++) {
			bucket_count.push_back(buckets[j].size());
			sink_bucket* bucket;
			CUDA_CHECK(hipMalloc(&bucket, sizeof(sink_bucket) * buckets[j].size()));
			CUDA_CHECK(hipMemcpyAsync(bucket, buckets[j].data(), sizeof(sink_bucket) * buckets[j].size(), hipMemcpyHostToDevice, stream));
			dev_buckets.push_back(bucket);
		}
		tm1.stop();
		PRINT("bucket time = %e\n", tm1.read());
		CUDA_CHECK(hipMemcpyAsync(params.bucket_cnt, bucket_count.data(), sizeof(int) * vol, hipMemcpyHostToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(params.buckets, dev_buckets.data(), sizeof(sink_bucket*) * vol, hipMemcpyHostToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(params.tree_neighbors, dev_tree_neighbors, sizeof(tree) * NCELLS * vol, hipMemcpyHostToDevice, stream));
		process_copies(std::move(copies), hipMemcpyHostToDevice, stream);
		CUDA_CHECK(hipStreamSynchronize(stream));
		tm.stop();
		PRINT("%e\n", tm.read());
		tm.reset();
		tm.start();
		PRINT("Launching kernel\n");
		CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dev_treepm_params), &params, sizeof(treepm_params)));
		kick_treepm_kernel<<<num_blocks,TREEPM_BLOCK_SIZE,0,stream>>>();

		count = 0;
		CUDA_CHECK(hipStreamSynchronize(stream));
		tm.stop();
		PRINT("%e\n", tm.read());
		tm.reset();
		tm.start();
		PRINT("Transfer back\n");
		copies.resize(0);
		count = 0;
		for (i[0] = box.begin[0]; i[0] != box.end[0]; i[0]++) {
			for (i[1] = box.begin[1]; i[1] != box.end[1]; i[1]++) {
				for (i[2] = box.begin[2]; i[2] != box.end[2]; i[2]++) {
					auto this_cell = chainmesh_get(i);
					const auto this_size = this_cell.pend - this_cell.pbegin;
					const auto begin = this_cell.pbegin;
					cpymem cpy;
					cpy.size = sizeof(float) * this_size;
					cpy.src = params.velx + count;
					cpy.dest = &particles_vel(XDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.vely + count;
					cpy.dest = &particles_vel(YDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.velz + count;
					cpy.dest = &particles_vel(ZDIM, begin);
					copies.push_back(cpy);
					cpy.size = sizeof(char) * this_size;
					cpy.src = params.rung + count;
					cpy.dest = &particles_rung(begin);
					copies.push_back(cpy);
#ifdef FORCE_TEST
					cpy.size = sizeof(float) * this_size;
					cpy.src = params.gx + count;
					cpy.dest = &particles_gforce(XDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.gy + count;
					cpy.dest = &particles_gforce(YDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.gz + count;
					cpy.dest = &particles_gforce(ZDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.pot + count;
					cpy.dest = &particles_pot(begin);
					copies.push_back(cpy);
#endif
					count += this_size;
				}
			}
		}
		process_copies(std::move(copies), hipMemcpyDeviceToHost, stream);
		CUDA_CHECK(hipStreamSynchronize(stream));
		params.free();
		free(dev_tree_neighbors);
		CUDA_CHECK(hipStreamDestroy(stream));
		tm.stop();
		for (int j = 0; j < vol; j++) {
			CUDA_CHECK(hipFree(dev_buckets[j]));
		}

		PRINT("%e\n", tm.read());
	}
}
