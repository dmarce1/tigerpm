#include "hip/hip_runtime.h"
#include <tigerpm/fmmpm.hpp>
#include <tigerpm/timer.hpp>
#include <tigerpm/options.hpp>
#include <tigerpm/particles.hpp>
#include <tigerpm/util.hpp>
#include <tigerpm/gravity_long.hpp>
#include <tigerpm/gravity_short.hpp>
#include <tigerpm/stack_vector.hpp>

#include <thrust/device_vector.h>

#define FMMPM_MIN_THREADS 16
#define FMMPM_BLOCK_SIZE 32

#include <algorithm>

__constant__ float rung_dt[MAX_RUNG] = { 1.0 / (1 << 0), 1.0 / (1 << 1), 1.0 / (1 << 2), 1.0 / (1 << 3), 1.0 / (1 << 4), 1.0 / (1 << 5), 1.0 / (1 << 6), 1.0
		/ (1 << 7), 1.0 / (1 << 8), 1.0 / (1 << 9), 1.0 / (1 << 10), 1.0 / (1 << 11), 1.0 / (1 << 12), 1.0 / (1 << 13), 1.0 / (1 << 14), 1.0 / (1 << 15), 1.0
		/ (1 << 16), 1.0 / (1 << 17), 1.0 / (1 << 18), 1.0 / (1 << 19), 1.0 / (1 << 20), 1.0 / (1 << 21), 1.0 / (1 << 22), 1.0 / (1 << 23), 1.0 / (1 << 24), 1.0
		/ (1 << 25), 1.0 / (1 << 26), 1.0 / (1 << 27), 1.0 / (1 << 28), 1.0 / (1 << 29), 1.0 / (1 << 30), 1.0 / (1 << 31) };

#define LIST_SIZE  (4*1024)
#define STACK_SIZE (32*1024)
#define MAX_DEPTH 64

struct checkitem {
	int index :24;
	int opened :8;
	tree* tr;

	CUDA_EXPORT inline
	bool is_leaf() const {
		return tr->is_leaf(index);
	}
	CUDA_EXPORT inline
	fixed32 get_x(int dim) const {
		return tr->get_x(dim, index);
	}
	CUDA_EXPORT inline
	float get_radius() const {
		return tr->get_radius(index);
	}
	CUDA_EXPORT inline
	array<checkitem, 2> get_children() {
		const auto indices = tr->get_children(index);
		array<checkitem, 2> c;
		c[0].index = indices[0];
		c[1].index = indices[1];
		c[0].opened = c[1].opened = 0;
	}
};

struct list_set {
	stack_vector<checkitem, STACK_SIZE, MAX_DEPTH> checklist;
	fixedcapvec<checkitem, LIST_SIZE> openlist;
	fixedcapvec<checkitem, LIST_SIZE> nextlist;
	fixedcapvec<checkitem, LIST_SIZE> multilist;
	fixedcapvec<checkitem, LIST_SIZE> partlist;
};

struct fmmpm_params {
	fixed32* x;
	fixed32* y;
	fixed32* z;
	float* velx;
	float* vely;
	float* velz;
	float* phi;
	char* rung;
	list_set* lists;
	tree* tree_neighbors;
	int* active;
	int min_rung;
	bool do_phi;
	float rs;
	float rcut;
	float GM;
	float eta;
	float t0;
	float scale;
	float hsoft;
	float inv2rs;
	float phi0;
	float twooversqrtpi;
	float h2;
	float hinv;
	float h2inv;
	float h3inv;
	float theta;
	int nsink_cells;
	bool first_call;
	int Nfour;
	range<int> phi_box;
#ifdef FORCE_TEST
	float* gx;
	float* gy;
	float* gz;
	float* pot;
#endif
	void allocate(size_t source_size, size_t sink_size, size_t cell_count, size_t big_cell_count, size_t phi_cell_count, int nblocks) {
		CUDA_CHECK(hipMalloc(&tree_neighbors, cell_count * NCELLS * sizeof(tree)));
		nsink_cells = cell_count;
		CUDA_CHECK(hipMalloc(&x, source_size * sizeof(fixed32)));
		CUDA_CHECK(hipMalloc(&y, source_size * sizeof(fixed32)));
		CUDA_CHECK(hipMalloc(&z, source_size * sizeof(fixed32)));
		CUDA_CHECK(hipMalloc(&velx, sink_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&vely, sink_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&velz, sink_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&lists, sizeof(list_set)));
		CUDA_CHECK(hipMalloc(&rung, sink_size * sizeof(char)));
		CUDA_CHECK(hipMalloc(&phi, phi_cell_count * sizeof(float)));
		CUDA_CHECK(hipMalloc(&active, nblocks * sizeof(int) * SINK_BUCKET_SIZE));
#ifdef FORCE_TEST
		CUDA_CHECK(hipMalloc(&gx, source_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&gy, source_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&gz, source_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&pot, source_size * sizeof(float)));
#endif
	}
	void free() {
		CUDA_CHECK(hipFree(x));
		CUDA_CHECK(hipFree(y));
		CUDA_CHECK(hipFree(z));
		CUDA_CHECK(hipFree(velx));
		CUDA_CHECK(hipFree(vely));
		CUDA_CHECK(hipFree(velz));
		CUDA_CHECK(hipFree(phi));
		CUDA_CHECK(hipFree(active));
		CUDA_CHECK(hipFree(rung));
		CUDA_CHECK(hipFree(lists));
		CUDA_CHECK(hipFree(tree_neighbors));
#ifdef FORCE_TEST
		CUDA_CHECK(hipFree(gx));
		CUDA_CHECK(hipFree(gy));
		CUDA_CHECK(hipFree(gz));
		CUDA_CHECK(hipFree(pot));
#endif
	}
};

static size_t mem_requirements(int nsources, int nsinks, int vol, int bigvol, int phivol) {
	size_t mem = 0;
	mem += NDIM * sizeof(fixed32) * nsources;
	mem += NDIM * sizeof(float) * nsinks;
	mem += sizeof(char) * nsinks;
	mem += NCELLS * bigvol * sizeof(tree);
	mem += 2 * sizeof(int) * vol;
	mem += sizeof(int) * vol;
	mem += sizeof(tree) * vol;
	mem += phivol * sizeof(float);
	mem += sizeof(fmmpm_params);
#ifdef FORCE_TEST
	mem += (NDIM + 1) * sizeof(float) * nsinks;
#endif
	return mem;
}

struct cpymem {
	void* dest;
	void* src;
	size_t size;
};

static void process_copies(vector<cpymem> copies, hipMemcpyKind direction, hipStream_t stream) {
	vector<cpymem> compressed;
	std::sort(copies.begin(), copies.end(), [](cpymem a, cpymem b) {
		return a.dest < b.dest;
	});
	for (int i = 0; i < copies.size(); i++) {
		cpymem copy = copies[i];
		for (int j = i + 1; j < copies.size(); j++) {
			if (((char*) copy.dest + copy.size == copies[j].dest) && ((char*) copy.src + copy.size == copies[j].src)) {
				copy.size += copies[j].size;
				i++;
			} else {
				break;
			}
		}
		compressed.push_back(copy);
	}
	PRINT("Compressed from %i to %i copies\n", copies.size(), compressed.size());
	for (int i = 0; i < compressed.size(); i++) {
		CUDA_CHECK(hipMemcpyAsync(compressed[i].dest, compressed[i].src, compressed[i].size, direction, stream));
	}
}

struct fmmpm_shmem {
	array<array<float, NDIM>, SINK_BUCKET_SIZE> g;
	array<float, SINK_BUCKET_SIZE> phi;
	array<fixed32, SINK_BUCKET_SIZE> x;
	array<fixed32, SINK_BUCKET_SIZE> y;
	array<fixed32, SINK_BUCKET_SIZE> z;
	array<fixed32, KICK_PP_MAX> srcx;
	array<fixed32, KICK_PP_MAX> srcy;
	array<fixed32, KICK_PP_MAX> srcz;
};

__constant__ fmmpm_params dev_fmmpm_params;

__device__ inline int compute_indices(int index, int& total) {
	const int& tid = threadIdx.x;
	for (int P = 1; P < FMMPM_BLOCK_SIZE; P *= 2) {
		auto tmp = __shfl_up_sync(0xFFFFFFFF, index, P);
		if (tid >= P) {
			index += tmp;
		}
	}
	total = __shfl_sync(0xFFFFFFFF, index, FMMPM_BLOCK_SIZE - 1);
	auto tmp = __shfl_up_sync(0xFFFFFFFF, index, 1);
	if (tid >= 1) {
		index = tmp;
	} else {
		index = 0;
	}
	return index;
}

template<class T>
__device__ inline void shared_reduce(T& number) {
	for (int P = warpSize / 2; P >= 1; P /= 2) {
		number += __shfl_xor_sync(0xffffffff, number, P);
	}
}

__device__ void pp_interactions() {

}

__device__ void pc_interactions() {

}

__device__ void cp_interactions() {

}

__device__ void cc_interactions() {

}

__device__ void do_kick(checkitem mycheck) {
	const int& tid = threadIdx.x;
	const int& bid = blockIdx.x;
	const int& gsz = gridDim.x;
	__shared__ extern int shmem_ptr[];
	fmmpm_shmem& shmem = (fmmpm_shmem&) (*shmem_ptr);
	const auto& params = dev_fmmpm_params;
	auto& checklist = params.lists->checklist;
	auto& multilist = params.lists->multilist;
	auto& partlist = params.lists->partlist;
	auto& openlist = params.lists->openlist;
	auto& nextlist = params.lists->nextlist;
	const auto& theta2inv = 1.0f / sqr(params.theta);
	const bool iamleaf = mycheck.is_leaf();
	const float myradius = mycheck.get_radius();
	const fixed32 sink_x = mycheck.get_x(XDIM);
	const fixed32 sink_y = mycheck.get_x(YDIM);
	const fixed32 sink_z = mycheck.get_x(ZDIM);
	do {
		bool multi = false;
		bool part = false;
		bool open = false;
		bool next = false;
		for (int ci = tid; ci < checklist.size(); ci += warpSize) {
			auto& check = checklist[ci];
			const fixed32 src_x = check.get_x(XDIM);
			const fixed32 src_y = check.get_x(YDIM);
			const fixed32 src_z = check.get_x(ZDIM);
			const float source_radius = check.get_radius();
			const bool source_isleaf = check.is_leaf();
			const float dx = distance(sink_x, src_x);
			const float dy = distance(sink_y, src_y);
			const float dz = distance(sink_z, src_z);
			const float R2 = sqr(dx, dy, dz);
			const bool far = R2 > sqr(myradius + source_radius) * theta2inv;
			if (far) {
				if (!check.opened) {
					multi = true;
				} else {
					part = true;
				}
			} else {
				if (source_isleaf) {
					if (check.opened) {
						part = true;
					} else {
						open = true;
						check.opened = 1;
					}
				} else {
					next = true;
				}
			}
			int index, total;
			index = multi;
			index = compute_indices(index, total) + multilist.size();
			__syncwarp();
			if (tid == 0) {
				multilist.resize(multilist.size() + total);
			}
			__syncwarp();
			multilist[index] = check;
			index = part;
			index = compute_indices(index, total) + partlist.size();
			__syncwarp();
			if (tid == 0) {
				multilist.resize(partlist.size() + total);
			}
			__syncwarp();
			partlist[index] = check;
			index = next;
			index = compute_indices(index, total) + nextlist.size();
			__syncwarp();
			if (tid == 0) {
				nextlist.resize(partlist.size() + total);
			}
			__syncwarp();
			nextlist[index] = check;
			index = next;
			index = compute_indices(index, total) + openlist.size();
			__syncwarp();
			if (tid == 0) {
				openlist.resize(partlist.size() + total);
			}
			__syncwarp();
			openlist[index] = check;
		}
		__syncwarp();
		if (tid == 0) {
			checklist.resize(nextlist.size() * 2);
		}
		__syncwarp();
		for (int ci = 0; ci < nextlist.size(); ci += warpSize) {
			const auto children = nextlist[ci].get_children();
			checklist[2 * ci] = children[0];
			checklist[2 * ci + 1] = children[1];
		}
		const int offset = checklist.size();
		__syncwarp();
		if (tid == 0) {
			checklist.resize(offset + openlist.size());
		}
		__syncwarp();
		for (int ci = 0; ci < openlist.size(); ci += warpSize) {
			checklist[ci + offset] = openlist[ci];
		}
		if( mycheck.opened == 0 ) {
			cc_interactions();
			cp_interactions();
		} else {
			pc_interactions();
			pp_interactions();
		}
		__syncwarp();
		if (tid == 0) {
			nextlist.resize(0);
			openlist.resize(0);
			multilist.resize(0);
			partlist.resize(0);
		}
		__syncwarp();
		if (iamleaf) {
			mycheck.opened = 1;
		}
	} while (iamleaf && checklist.size());
}

__global__ void kick_fmmpm_kernel() {
	const int& tid = threadIdx.x;
	const int& bid = blockIdx.x;
	const int& gsz = gridDim.x;
	__shared__ extern int shmem_ptr[];
	fmmpm_shmem& shmem = (fmmpm_shmem&) (*shmem_ptr);
	const auto& params = dev_fmmpm_params;
	new (params.lists) list_set();
	auto& checklist = params.lists->checklist;
	const int cell_begin = size_t(bid) * (size_t) params.nsink_cells / (size_t) gsz;
	const int cell_end = size_t(bid + 1) * (size_t) params.nsink_cells / (size_t) gsz;
	for (int cell_index = cell_begin; cell_index < cell_end; cell_index++) {
		if (tid == 0) {
			checklist.resize(NCELLS);
		}
		__syncwarp();
		for (int treei = tid; treei < NCELLS; treei += warpSize) {
			checklist[treei].tr = params.tree_neighbors + cell_index * NCELLS + treei;
			checklist[treei].opened = 0;
			checklist[treei].index = 0;
		}
		checkitem mycheck;
		mycheck.tr = params.tree_neighbors + cell_index * NCELLS + NCELLS / 2;
		mycheck.opened = 0;
		mycheck.index = 0;
		__syncwarp();
		do_kick(mycheck);
	}

	params.lists->~list_set();
}

void kick_fmmpm(vector<tree> trees, range<int> box, int min_rung, double scale, double t0, bool first_call) {
	PRINT("shmem size = %i\n", sizeof(fmmpm_shmem));
//hipFuncCache_t pCacheConfig;
	hipDeviceSetCacheConfig (hipFuncCachePreferShared);
//	hipDeviceGetCacheConfig(&pCacheConfig);
	timer tm;
	size_t nsources = 0;
	size_t nsinks = 0;
	array<int, NDIM> i;
	const auto bigbox = box.pad(1);
	const size_t bigvol = bigbox.volume();
	const size_t vol = box.volume();
	int tree_size = 0;
	print("%i\n", bigvol);
	for (i[0] = bigbox.begin[0]; i[0] != bigbox.end[0]; i[0]++) {
		for (i[1] = bigbox.begin[1]; i[1] != bigbox.end[1]; i[1]++) {
			for (i[2] = bigbox.begin[2]; i[2] != bigbox.end[2]; i[2]++) {
				auto this_cell = chainmesh_get(i);
				nsources += this_cell.pend - this_cell.pbegin;
				const int index = bigbox.index(i);
				tree_size += trees[index].size() * sizeof(tree_node) + sizeof(tree);
			}
		}
	}
	PRINT("tree size = %e GB\n", tree_size / 1024 / 1024 / 1024.0);
	auto phibox = box;
	for (int dim = 0; dim < NDIM; dim++) {
		phibox.begin[dim] *= get_options().four_o_chain;
		phibox.end[dim] *= get_options().four_o_chain;
	}
	phibox = phibox.pad(PHI_BW);
	hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(kick_fmmpm_kernel));
	if (attr.maxThreadsPerBlock < FMMPM_BLOCK_SIZE) {
		PRINT("This CUDA device will not run kick_pme_kernel with the required number of threads (%i)\n", FMMPM_BLOCK_SIZE);
		abort();
	}
	int occupancy;
	CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor ( &occupancy, kick_fmmpm_kernel,FMMPM_BLOCK_SIZE, sizeof(fmmpm_shmem)));
	PRINT("Occupancy = %i\n", occupancy);
	int num_blocks = 2 * occupancy * cuda_smp_count();
	const size_t mem_required = mem_requirements(nsources, nsinks, vol, bigvol, phibox.volume()) + tree_size + sizeof(fmmpm_params);
	const size_t free_mem = (size_t) 85 * cuda_free_mem() / size_t(100);
	PRINT("required = %li freemem = %li\n", mem_required, free_mem);
	if (mem_required > free_mem) {
		const auto child_boxes = box.split();
		PRINT("Splitting\n");
		kick_fmmpm(trees, child_boxes.first, min_rung, scale, t0, first_call);
		kick_fmmpm(std::move(trees), child_boxes.second, min_rung, scale, t0, first_call);
	} else {
		cuda_set_device();
		PRINT("Data transfer\n");
		tm.start();
		fmmpm_params params;
		params.allocate(nsources, nsinks, vol, bigvol, phibox.volume(), num_blocks);
		tm.stop();
		PRINT("%e\n", tm.read());
		tm.start();
		params.theta = 0.8;
		params.min_rung = min_rung;
		params.rs = get_options().rs;
		params.do_phi = true;
		params.rcut = 1.0 / get_options().chain_dim;
		params.hsoft = get_options().hsoft;
		params.phi0 = std::pow(get_options().parts_dim, NDIM) * 4.0 * M_PI * sqr(params.rs) - SELF_PHI / params.hsoft;
		PRINT("RCUT = %e RS\n", params.rcut / params.rs);
		params.GM = get_options().GM;
		params.Nfour = get_options().four_dim;
		params.phi_box = phibox;
		params.eta = get_options().eta;
		params.first_call = first_call;
		params.t0 = t0;
		params.scale = scale;
		params.inv2rs = 1.0f / params.rs / 2.0f;
		params.twooversqrtpi = 2.0f / sqrtf(M_PI);
		params.h2 = sqr(params.hsoft);
		params.hinv = 1.f / params.hsoft;
		params.h2inv = sqr(params.hinv);
		params.h3inv = params.hinv * sqr(params.hinv);
		tree* dev_tree_neighbors = (tree*) malloc(sizeof(tree) * NCELLS * vol);
		hipStream_t stream;
		CUDA_CHECK(hipStreamCreate(&stream));
		auto phi = gravity_long_get_phi(phibox);
		CUDA_CHECK(hipMemcpyAsync(params.phi, phi.data(), sizeof(float) * phi.size(), hipMemcpyHostToDevice, stream));

		struct cell_data {
			int box_index;
			int bigbox_index;
			chaincell cell;
		};
		vector<cell_data> chaincells;
		for (i[0] = bigbox.begin[0]; i[0] != bigbox.end[0]; i[0]++) {
			for (i[1] = bigbox.begin[1]; i[1] != bigbox.end[1]; i[1]++) {
				for (i[2] = bigbox.begin[2]; i[2] != bigbox.end[2]; i[2]++) {
					cell_data entry;
					entry.bigbox_index = bigbox.index(i);
					entry.cell = chainmesh_get(i);
					if (box.contains(i)) {
						const int q = box.index(i);
						entry.box_index = q;
					} else {
						entry.box_index = -1;
					}
					chaincells.push_back(entry);
				}
			}
		}
		std::sort(chaincells.begin(), chaincells.end(), [](cell_data a, cell_data b) {
			return a.cell.pbegin < b.cell.pbegin;
		});
		size_t count = 0;
		vector<cpymem> copies;
		for (int j = 0; j < chaincells.size(); j++) {
			auto this_cell = chaincells[j].cell;
			const auto this_size = this_cell.pend - this_cell.pbegin;
			const auto begin = this_cell.pbegin;
			const auto dif = count - begin;
			const int l = chaincells[j].bigbox_index;
			trees[l].adjust_indexes(dif);
			cpymem cpy;
			cpy.size = sizeof(fixed32) * this_size;
			cpy.dest = params.x + count;
			cpy.src = &particles_pos(XDIM, begin);
			copies.push_back(cpy);
			cpy.dest = params.y + count;
			cpy.src = &particles_pos(YDIM, begin);
			copies.push_back(cpy);
			cpy.dest = params.z + count;
			cpy.src = &particles_pos(ZDIM, begin);
			copies.push_back(cpy);
			count += this_size;
		}

		vector<tree_node> all_trees;
		tree_node* dev_all_trees;
		count = 0;
		size_t trees_size = 0;
		for (int j = 0; j < bigvol; j++) {
			trees_size += trees[j].size();
		}
		all_trees.resize(trees_size);
		CUDA_CHECK(hipMallocAsync(&dev_all_trees, sizeof(tree_node) * trees_size, stream));
		count = 0;
		vector<tree> dev_trees(bigvol);
		for (int j = 0; j < bigvol; j++) {
			dev_trees[j] = trees[j].to_device();
			dev_trees[j].nodes = dev_all_trees + count;
			std::memcpy(all_trees.data() + count, trees[j].nodes, sizeof(tree_node) * trees[j].size());
			count += dev_trees[j].size();
		}
		CUDA_CHECK(hipMemcpyAsync(dev_all_trees, all_trees.data(), trees_size * sizeof(tree_node), hipMemcpyHostToDevice));
		count = 0;
		for (i[0] = box.begin[0]; i[0] != box.end[0]; i[0]++) {
			for (i[1] = box.begin[1]; i[1] != box.end[1]; i[1]++) {
				for (i[2] = box.begin[2]; i[2] != box.end[2]; i[2]++) {
					auto this_cell = chainmesh_get(i);
					const auto this_size = this_cell.pend - this_cell.pbegin;
					const auto begin = this_cell.pbegin;
					cpymem cpy;
					const int l = box.index(i);
					cpy.size = sizeof(float) * this_size;
					cpy.dest = params.velx + count;
					cpy.src = &particles_vel(XDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.vely + count;
					cpy.src = &particles_vel(YDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.velz + count;
					cpy.src = &particles_vel(ZDIM, begin);
					copies.push_back(cpy);
					cpy.size = sizeof(char) * this_size;
					cpy.dest = params.rung + count;
					cpy.src = &particles_rung(begin);
					copies.push_back(cpy);
#ifdef FORCE_TEST
					cpy.size = sizeof(float) * this_size;
					cpy.dest = params.gx + count;
					cpy.src = &particles_gforce(XDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.gy + count;
					cpy.src = &particles_gforce(YDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.gz + count;
					cpy.src = &particles_gforce(ZDIM, begin);
					copies.push_back(cpy);
					cpy.dest = params.pot + count;
					cpy.src = &particles_pot(begin);
					copies.push_back(cpy);
#endif
					count += this_size;
					array<int, NDIM> j;
					int p = 0;
					for (j[0] = i[0] - 1; j[0] <= i[0] + 1; j[0]++) {
						for (j[1] = i[1] - 1; j[1] <= i[1] + 1; j[1]++) {
							for (j[2] = i[2] - 1; j[2] <= i[2] + 1; j[2]++) {
								const int k = bigbox.index(j);
								std::memcpy(&dev_tree_neighbors[p + NCELLS * l], &dev_trees[k], sizeof(tree));
								p++;
							}
						}
					}
				}
			}
		}
		CUDA_CHECK(hipMemcpyAsync(params.tree_neighbors, dev_tree_neighbors, sizeof(tree) * NCELLS * vol, hipMemcpyHostToDevice, stream));
		process_copies(std::move(copies), hipMemcpyHostToDevice, stream);
		CUDA_CHECK(hipStreamSynchronize(stream));
		tm.stop();
		PRINT("Transfer time %e\n", tm.read());
		tm.reset();
		tm.start();
		PRINT("Launching kernel\n");
		CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dev_fmmpm_params), &params, sizeof(fmmpm_params)));
		kick_fmmpm_kernel<<<num_blocks,FMMPM_BLOCK_SIZE,sizeof(fmmpm_shmem),stream>>>();

		count = 0;
		CUDA_CHECK(hipStreamSynchronize(stream));
		tm.stop();
		PRINT("%e\n", tm.read());
		tm.reset();
		tm.start();
		PRINT("Transfer back\n");
		copies.resize(0);
		count = 0;
		for (i[0] = box.begin[0]; i[0] != box.end[0]; i[0]++) {
			for (i[1] = box.begin[1]; i[1] != box.end[1]; i[1]++) {
				for (i[2] = box.begin[2]; i[2] != box.end[2]; i[2]++) {
					auto this_cell = chainmesh_get(i);
					const auto this_size = this_cell.pend - this_cell.pbegin;
					const auto begin = this_cell.pbegin;
					cpymem cpy;
					cpy.size = sizeof(float) * this_size;
					cpy.src = params.velx + count;
					cpy.dest = &particles_vel(XDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.vely + count;
					cpy.dest = &particles_vel(YDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.velz + count;
					cpy.dest = &particles_vel(ZDIM, begin);
					copies.push_back(cpy);
					cpy.size = sizeof(char) * this_size;
					cpy.src = params.rung + count;
					cpy.dest = &particles_rung(begin);
					copies.push_back(cpy);
#ifdef FORCE_TEST
					cpy.size = sizeof(float) * this_size;
					cpy.src = params.gx + count;
					cpy.dest = &particles_gforce(XDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.gy + count;
					cpy.dest = &particles_gforce(YDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.gz + count;
					cpy.dest = &particles_gforce(ZDIM, begin);
					copies.push_back(cpy);
					cpy.src = params.pot + count;
					cpy.dest = &particles_pot(begin);
					copies.push_back(cpy);
#endif
					count += this_size;
				}
			}
		}
		process_copies(std::move(copies), hipMemcpyDeviceToHost, stream);
		CUDA_CHECK(hipStreamSynchronize(stream));
		params.free();
		free(dev_tree_neighbors);
		CUDA_CHECK(hipStreamDestroy(stream));
		CUDA_CHECK(hipFree(dev_all_trees));
		tm.stop();
	}
}
