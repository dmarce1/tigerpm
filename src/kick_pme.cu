#include <tigerpm/kick_pme.hpp>
#include <tigerpm/range.hpp>
#include <tigerpm/chainmesh.hpp>
#include <tigerpm/particles.hpp>

#define NCELLS 27

struct source_cell {
	int begin;
	int end;
};

struct sink_cell {
	int begin;
	int end;
};

static size_t mem_requirements(range<int> box) {
	size_t mem = 0;
	const auto bigbox = box.pad(1);
	mem += NDIM * sizeof(fixed32) * bigbox.volume();
	mem += NDIM * sizeof(float) * box.volume();
	mem += sizeof(char) * box.volume();
	mem += NCELLS * bigbox.volume() * sizeof(source_cell);
	mem += box.volume() * sizeof(sink_cell);
#ifdef TEST_FORCE
	mem += (NDIM+1) * sizeof(float) * box.volume();
#endif
	return mem;
}

struct kick_pme_kernel_params {
	fixed32* x;
	fixed32* y;
	fixed32* z;
	float* velx;
	float* vely;
	float* velz;
	char* rung;
	source_cell* source_cells;
	sink_cell* sink_cells;
#ifdef TEST_FORCE
	float* gx;
	float* gy;
	float* gz;
	float* pot;
#endif
	void allocate(size_t source_size, size_t sink_size, size_t cell_count) {
		CUDA_CHECK(hipMalloc(&x, source_size * sizeof(fixed32)));
		CUDA_CHECK(hipMalloc(&y, source_size * sizeof(fixed32)));
		CUDA_CHECK(hipMalloc(&z, source_size * sizeof(fixed32)));
		CUDA_CHECK(hipMalloc(&velx, source_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&vely, source_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&velz, source_size * sizeof(float)));
		CUDA_CHECK(hipMalloc(&rung, source_size * sizeof(char)));
		CUDA_CHECK(hipMalloc(&source_cells, cell_count * NCELLS * sizeof(source_cell)));
		CUDA_CHECK(hipMalloc(&sink_cells, cell_count * sizeof(source_cell)));
#ifdef TEST_FORCE
		CUDA_CHECK(hipMalloc(&gx,source_size*sizeof(float)));
		CUDA_CHECK(hipMalloc(&gy,source_size*sizeof(float)));
		CUDA_CHECK(hipMalloc(&gz,source_size*sizeof(float)));
		CUDA_CHECK(hipMalloc(&pot,source_size*sizeof(float)));
#endif
	}
	void free() {
		CUDA_CHECK(hipFree(x));
		CUDA_CHECK(hipFree(y));
		CUDA_CHECK(hipFree(z));
		CUDA_CHECK(hipFree(velx));
		CUDA_CHECK(hipFree(vely));
		CUDA_CHECK(hipFree(velz));
		CUDA_CHECK(hipFree(rung));
		CUDA_CHECK(hipFree(source_cells));
		CUDA_CHECK(hipFree(sink_cells));
#ifdef TEST_FORCE
		CUDA_CHECK(hipFree(gx));
		CUDA_CHECK(hipFree(gy));
		CUDA_CHECK(hipFree(gz));
		CUDA_CHECK(hipFree(pot));
#endif
	}
};

void kick_pme(range<int> box) {
	const size_t mem_required = mem_requirements(box);
	if (mem_required > cuda_free_mem() * 85 / 100) {
		const auto child_boxes = box.split();
		kick_pme(child_boxes.first);
		kick_pme(child_boxes.second);
	} else {
		cuda_set_device();
		const auto bigbox = box.pad(1);
		const size_t bigvol = bigbox.volume();
		const size_t vol = box.volume();
		kick_pme_kernel_params params;
		size_t nsources = 0;
		size_t nsinks = 0;
		array<int, NDIM> i;
		for (i[0] = bigbox.begin[0]; i[0] != bigbox.end[0]; i[0]++) {
			for (i[1] = bigbox.begin[1]; i[1] != bigbox.end[1]; i[1]++) {
				for (i[2] = bigbox.begin[2]; i[2] != bigbox.end[2]; i[2]++) {
					auto this_cell = chainmesh_get(i);
					nsources += this_cell.pend - this_cell.pbegin;
				}
			}
		}
		for (i[0] = box.begin[0]; i[0] != box.end[0]; i[0]++) {
			for (i[1] = box.begin[1]; i[1] != box.end[1]; i[1]++) {
				for (i[2] = box.begin[2]; i[2] != box.end[2]; i[2]++) {
					auto this_cell = chainmesh_get(i);
					nsinks += this_cell.pend - this_cell.pbegin;
				}
			}
		}
		params.allocate(nsources, nsinks, vol);
		std::vector<source_cell> source_cells(bigvol);
		std::vector<source_cell> dev_source_cells(NCELLS * vol);
		std::vector<sink_cell> sink_cells(bigvol);
		size_t count = 0;
		hipStream_t stream;
		hipStreamCreate(&stream);
		for (i[0] = bigbox.begin[0]; i[0] != bigbox.end[0]; i[0]++) {
			for (i[1] = bigbox.begin[1]; i[1] != bigbox.end[1]; i[1]++) {
				for (i[2] = bigbox.begin[2]; i[2] != bigbox.end[2]; i[2]++) {
					auto this_cell = chainmesh_get(i);
					const auto this_size = this_cell.pend - this_cell.pbegin;
					const auto begin = this_cell.pbegin;
					CUDA_CHECK(
							hipMemcpyAsync(params.x + count, &particles_pos(XDIM, begin), sizeof(fixed32) * this_size, hipMemcpyHostToDevice, stream));
					CUDA_CHECK(
							hipMemcpyAsync(params.y + count, &particles_pos(YDIM, begin), sizeof(fixed32) * this_size, hipMemcpyHostToDevice, stream));
					CUDA_CHECK(
							hipMemcpyAsync(params.z + count, &particles_pos(ZDIM, begin), sizeof(fixed32) * this_size, hipMemcpyHostToDevice, stream));
					const int l = bigbox.index(i);
					source_cells[l].begin = count;
					count += this_size;
					source_cells[l].end = count;
				}
			}
		}
		count = 0;
		for (i[0] = box.begin[0]; i[0] != box.end[0]; i[0]++) {
			for (i[1] = box.begin[1]; i[1] != box.end[1]; i[1]++) {
				for (i[2] = box.begin[2]; i[2] != box.end[2]; i[2]++) {
					auto this_cell = chainmesh_get(i);
					const auto this_size = this_cell.pend - this_cell.pbegin;
					const auto begin = this_cell.pbegin;
					CUDA_CHECK(
							hipMemcpyAsync(params.velx + count, &particles_vel(XDIM, begin), sizeof(float) * this_size, hipMemcpyHostToDevice, stream));
					CUDA_CHECK(
							hipMemcpyAsync(params.vely + count, &particles_vel(YDIM, begin), sizeof(float) * this_size, hipMemcpyHostToDevice, stream));
					CUDA_CHECK(
							hipMemcpyAsync(params.velz + count, &particles_vel(ZDIM, begin), sizeof(float) * this_size, hipMemcpyHostToDevice, stream));
					CUDA_CHECK(
							hipMemcpyAsync(params.rung + count, &particles_rung(begin), sizeof(char) * this_size,
									hipMemcpyHostToDevice, stream));
#ifdef TEST_FORCE
					CUDA_CHECK(
							hipMemcpyAsync(params.gx + count, &particles_gforce(XDIM, begin), sizeof(float) * this_size, hipMemcpyHostToDevice, stream));
					CUDA_CHECK(
							hipMemcpyAsync(params.gy + count, &particles_gforce(YDIM, begin), sizeof(float) * this_size, hipMemcpyHostToDevice, stream));
					CUDA_CHECK(
							hipMemcpyAsync(params.gz + count, &particles_gforce(ZDIM, begin), sizeof(float) * this_size, hipMemcpyHostToDevice, stream));
					CUDA_CHECK(
							hipMemcpyAsync(params.pot + count, &particles_pot(begin), sizeof(float) * this_size, hipMemcpyHostToDevice, stream));
#endif
					const int l = box.index(i);
					sink_cells[l].begin = count;
					count += this_size;
					sink_cells[l].end = count;
					array<int, NDIM> j;
					int p = 0;
					for (j[0] = i[0] - 1; j[0] <= i[0] + 1; j[0]++) {
						for (j[1] = i[1] - 1; j[1] <= i[1] + 1; j[1]++) {
							for (j[2] = i[1] - 1; j[1] <= i[2] + 1; j[2]++) {
								const int k = bigbox.index(j);
								dev_source_cells[p + NCELLS * l] = source_cells[k];
								p++;
							}
						}
					}

				}
			}
		}
		CUDA_CHECK(
				hipMemcpyAsync(params.sink_cells, sink_cells.data(), sizeof(sink_cell) * sink_cells.size(),
						hipMemcpyHostToDevice));
		CUDA_CHECK(
				hipMemcpyAsync(params.source_cells, dev_source_cells.data(), sizeof(source_cell) * dev_source_cells.size(),
						hipMemcpyHostToDevice));









		count = 0;
		for (i[0] = box.begin[0]; i[0] != box.end[0]; i[0]++) {
			for (i[1] = box.begin[1]; i[1] != box.end[1]; i[1]++) {
				for (i[2] = box.begin[2]; i[2] != box.end[2]; i[2]++) {
					auto this_cell = chainmesh_get(i);
					const auto this_size = this_cell.pend - this_cell.pbegin;
					const auto begin = this_cell.pbegin;
					CUDA_CHECK(
							hipMemcpyAsync(&particles_vel(XDIM, begin), params.velx + count,sizeof(float) * this_size, hipMemcpyDeviceToHost, stream));
					CUDA_CHECK(
							hipMemcpyAsync(&particles_vel(YDIM, begin), params.vely + count, sizeof(float) * this_size, hipMemcpyDeviceToHost, stream));
					CUDA_CHECK(
							hipMemcpyAsync(&particles_vel(ZDIM, begin), params.velz + count, sizeof(float) * this_size, hipMemcpyDeviceToHost, stream));
					CUDA_CHECK(
							hipMemcpyAsync(&particles_rung(begin), params.rung + count, sizeof(char) * this_size,
									hipMemcpyDeviceToHost));
#ifdef TEST_FORCE
					CUDA_CHECK(
							hipMemcpyAsync(&particles_gforce(XDIM, begin), params.gx + count, sizeof(float) * this_size, hipMemcpyDeviceToHost, stream));
					CUDA_CHECK(
							hipMemcpyAsync(&particles_gforce(YDIM, begin), params.gy + count, sizeof(float) * this_size, hipMemcpyDeviceToHost, stream));
					CUDA_CHECK(
							hipMemcpyAsync(&particles_gforce(ZDIM, begin), params.gz + count, sizeof(float) * this_size, hipMemcpyDeviceToHost, stream));
					CUDA_CHECK(
							hipMemcpyAsync(&particles_pot(begin), params.pot + count, sizeof(float) * this_size, hipMemcpyDeviceToHost, stream));
#endif
				}
			}
		}

		params.free();
		hipStreamDestroy(stream);
	}
}
